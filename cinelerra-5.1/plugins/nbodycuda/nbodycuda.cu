#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"
#include "helper_gl.h"

#include "nbodycuda.h"

void N_BodyCuda::init()
{
	checkCudaErrors(hipEventCreate(&startEvent));
	checkCudaErrors(hipEventCreate(&stopEvent));
	checkCudaErrors(hipEventCreate(&hostMemSyncEvent));
}

void N_BodyCuda::init_dev()
{
//	int dev_id = findCudaDevice(argc, (const char **)argv);
	int dev_id = gpuGetMaxGflopsDeviceId();
	checkCudaErrors(hipSetDevice(dev_id));
	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev_id));
printf("GPU Device %d: \"%s\" with compute capability %d.%d\n",
  dev_id, deviceProp.name, deviceProp.major, deviceProp.minor);
	version = deviceProp.major * 10 + deviceProp.minor;
	numSMs = deviceProp.multiProcessorCount;
}


void N_BodyCuda::finish()
{
	checkCudaErrors(hipEventDestroy(startEvent));
	checkCudaErrors(hipEventDestroy(stopEvent));
	checkCudaErrors(hipEventDestroy(hostMemSyncEvent));
}

