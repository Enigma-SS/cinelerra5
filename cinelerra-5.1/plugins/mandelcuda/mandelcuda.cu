#include "hip/hip_runtime.h"
#include "mandelcuda.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "hip/hip_runtime_api.h"
#include "helper_gl.h"

// The dimensions of the thread block
#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16
#define ABS(n) ((n) < 0 ? -(n) : (n))

void MandelCuda::init_dev()
{
	if( numSMs ) return;
//	int dev_id = findCudaDevice(argc, (const char **)argv);
	int dev_id = gpuGetMaxGflopsDeviceId();
	checkCudaErrors(hipSetDevice(dev_id));
	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev_id));
printf("GPU Device %d: \"%s\" with compute capability %d.%d\n",
  dev_id, deviceProp.name, deviceProp.major, deviceProp.minor);
	version = deviceProp.major * 10 + deviceProp.minor;
	numSMs = deviceProp.multiProcessorCount;
	if( !numSMs ) numSMs = -1;
}

void MandelCuda::init(int pbo, int pw, int ph)
{
	if( pbo_id >= 0 ) return;
	pbo_id = pbo;  pbo_w = pw;  pbo_h = ph;
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pbo, pbo_id, cudaGraphicsMapFlagsNone));
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo, 0));
	size_t pbo_bytes = 0;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer(&pbo_mem, &pbo_bytes, cuda_pbo));
}

void MandelCuda::finish()
{
	pbo_id = -1;
	pbo_w = pbo_h = 0;
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo));
	pbo_mem = 0;
	hipGraphicsUnregisterResource(cuda_pbo);  cuda_pbo = 0;
}


MandelCuda::MandelCuda()
{
	version = 0;
	numSMs = 0;
	pbo_id = -1;
	pbo_w = pbo_h = 0;
	cuda_pbo = 0;
	pbo_mem = 0;
}
MandelCuda::~MandelCuda()
{
}

static inline int iDivUp(int a, int b)
{
    int v = a / b;
    return a % b ? v+1 : v;
}

// Determine if two pixel colors are within tolerance
__device__ inline int CheckColors(const uchar4 &color0, const uchar4 &color1)
{
	int x = color1.x - color0.x;
	if( ABS(x) > 10 ) return 1;
	int y = color1.y - color0.y;
        if( ABS(y) > 10 ) return 1;
	int z = color1.z - color0.z;
        if( ABS(z) > 10 ) return 1;
	return 0;
}


// The core MandelCuda calculation function template
template<class T> __device__
inline int CalcCore(const int n, T ix, T iy, T xC, T yC)
{
    T x = ix, y = iy;
    T xx = x * x, yy = y * y;
    int i = n;
    while( --i && (xx + yy < 4.0f) ) {
    	y = x * y +  x * y + yC ;  // 2*x*y + yC
        x = xx - yy + xC ;
        yy = y * y;
        xx = x * x;
    }

    return i;
}

template<class T> __global__
void Calc(uchar4 *dst, const int img_w, const int img_h, const int is_julia,
		const int crunch, const int gridWidth, const int numBlocks,
		const T x_off, const T y_off, const T x_julia, const T y_julia, const T scale,
		const uchar4 colors, const int frame, const int animationFrame)
{
	// loop until all blocks completed
	for( unsigned int bidx=blockIdx.x; bidx<numBlocks; bidx+=gridDim.x ) {
		unsigned int blockX = bidx % gridWidth;
		unsigned int blockY = bidx / gridWidth;
		const int x = blockDim.x * blockX + threadIdx.x;
		const int y = blockDim.y * blockY + threadIdx.y;
		if( x >= img_w || y >= img_h ) continue;
		int pi = img_w*y + x, n = !frame ? 1 : 0;
		uchar4 pixel = dst[pi];
		if( !n && x > 0 )
			n += CheckColors(pixel, dst[pi-1]);
		if( !n && x+1 < img_w )
			n += CheckColors(pixel, dst[pi+1]);
		if( !n && y > 0 )
			n += CheckColors(pixel, dst[pi-img_w]);
		if( !n && y+1 < img_h )
			n += CheckColors(pixel, dst[pi+img_w]);
		if( !n ) continue;

		const T tx = T(x) * scale + x_off;
		const T ty = T(y) * scale + y_off;
		const T ix = is_julia ? tx : 0;
		const T iy = is_julia ? ty : 0;
		const T xC = is_julia ? x_julia : tx;
		const T yC = is_julia ? y_julia : ty;
		int m = CalcCore(crunch, ix,iy, xC,yC);
		m = m > 0 ? crunch - m : 0;
		if( m ) m += animationFrame;

		uchar4 color;
		color.x = m * colors.x;
		color.y = m * colors.y;
		color.z = m * colors.z;
		color.w = 0;

		int frame1 = frame+1, frame2 = frame1/2;
		color.x = (pixel.x * frame + color.x + frame2) / frame1;
		color.y = (pixel.y * frame + color.y + frame2) / frame1;
		color.z = (pixel.z * frame + color.z + frame2) / frame1;
		dst[pi] = color; // Output the pixel
	}
}


void MandelCuda::Run(unsigned char *data, unsigned int size, int is_julia, int crunch,
		double x_off, double y_off, double x_julia, double y_julia, double scale,
		uchar4 colors, int pass, int animationFrame)
{
	if( numSMs < 0 ) return;
	checkCudaErrors(hipMemcpy(pbo_mem, data, size, hipMemcpyHostToDevice));
	dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 grid(iDivUp(pbo_w, BLOCKDIM_X), iDivUp(pbo_h, BLOCKDIM_Y));
	Calc<float><<<numSMs, threads>>>((uchar4 *)pbo_mem, pbo_w, pbo_h,
			is_julia, crunch, grid.x, grid.x*grid.y,
			float(x_off), float(y_off), float(x_julia), float(y_julia), float(scale),
			colors, pass, animationFrame);
	checkCudaErrors(hipMemcpy(data, pbo_mem, size, hipMemcpyDeviceToHost));
}

